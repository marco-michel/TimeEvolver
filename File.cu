#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_complex.h>

#include "Header.cuh"

// Kernel to compute negativeH = -1.0 * d_HRet_values[j-1+j*m]
// The result is stored in d_negativeH[0]
__global__ void computeNegative(const hipDoubleComplex* d_HRet_values, hipDoubleComplex* d_negativeH, int idx)
{
    // Use a single thread to perform the computation.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_negativeH[0] = hipCmul(make_hipDoubleComplex(-1.0, 0.0), d_HRet_values[idx]);
    }
}



// Exposed interface function using C linkage
extern "C" void launchComputeNegative(const hipDoubleComplex* d_HRet_values,
    hipDoubleComplex* d_negativeH, int idx)
{
    // Launch the kernel
    computeNegative << <1, 1 >> > (d_HRet_values, d_negativeH, idx);
    // Optionally, synchronize to wait for kernel completion
    hipDeviceSynchronize();
}
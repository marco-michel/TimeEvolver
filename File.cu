#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_complex.h>

#include "Header.cuh"

// Kernel to compute negativeH = -1.0 * d_HRet_values[j-1+j*m]
// The result is stored in d_negativeH[0]
__global__ void computeNegative(const hipDoubleComplex* d_HRet_values, hipDoubleComplex* d_negativeH, int idx)
{
    // Use a single thread to perform the computation.
    if (threadIdx.x == 0 && blockIdx.x == 0) {
        d_negativeH[0] = hipCmul(make_hipDoubleComplex(-1.0, 0.0), d_HRet_values[idx]);
    }
}

__global__ void computeFourOutputsFromDouble(const double* input,
    hipDoubleComplex* out0,
    hipDoubleComplex* out1,
    hipDoubleComplex* out2,
    hipDoubleComplex* out3)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        double d = *input;  // Read the input value.

        // Convert input to a complex number.
        hipDoubleComplex cVal = make_hipDoubleComplex(d, 0.0);

        // Write the two outputs that are just the input value.
        *out0 = cVal;
        *out1 = cVal;

        // Compute and write the inverse: 1/d.
        *out2 = make_hipDoubleComplex(1.0 / d, 0.0);

        // Compute and write the negative: -d.
        *out3 = make_hipDoubleComplex(-d, 0.0);
    }
}

// Wrapper function callable from standard C++ code.
extern "C" void launchComputeFourOutputsFromDouble(const double* d_input,
    hipDoubleComplex* d_out0,
    hipDoubleComplex* d_out1,
    hipDoubleComplex* d_out2,
    hipDoubleComplex* d_out3)
{
    // Launch the kernel with one block and one thread.
    computeFourOutputsFromDouble << <1, 1 >> > (d_input, d_out0, d_out1, d_out2, d_out3);
    hipDeviceSynchronize();
}


// Exposed interface function using C linkage
extern "C" void launchComputeNegative(const hipDoubleComplex* d_HRet_values,
    hipDoubleComplex* d_negativeH, int idx)
{
    // Launch the kernel
    computeNegative << <1, 1 >> > (d_HRet_values, d_negativeH, idx);
    // Optionally, synchronize to wait for kernel completion
    hipDeviceSynchronize();
}